
#include <hip/hip_runtime.h>
#define THREADS_PER_BLOCK 3
#define TOTAL_BLOCKS 1

__global__ void dot_product (int* a, int*b, int*c)
{
     __shared__ int multiplicationStorage [THREADS_PER_BLOCK]; 

     multiplicationStorage[threadIdx.x] = a[threadIdx.x] * b[threadIdx.x];

	__syncthreads(); 

     if (threadIdx.x == 0){
	//compute sum
	int tempSum = 0; 
	for (int i = 0; i < THREADS_PER_BLOCK; i++){
		tempSum+=multiplicationStorage[i];
	}
	*c = tempSum; 
	//atomicAdd(c,tempSum); 
     }
	
}
