#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_complex.h>


static const int WORK_SIZE = 10;

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }

int main(void) {
	int *d = NULL;
	int i;
	float2 idata[WORK_SIZE];
	float2 odata[WORK_SIZE];

	for (i = 0; i < WORK_SIZE; i++){
		idata[i].x = i;
		idata[i].y = 0;
	}

		hipfftHandle plan;
		hipfftComplex *data;
		hipMalloc((void**)&data, sizeof(float2)*WORK_SIZE);
		hipMemcpy(data,idata,sizeof(float2)*WORK_SIZE,hipMemcpyHostToDevice);
		hipfftPlan1d(&plan, WORK_SIZE, HIPFFT_C2C,1);
		hipfftExecC2C(plan, data, data, HIPFFT_FORWARD);

		hipDeviceSynchronize();



	CUDA_CHECK_RETURN(hipMemcpy(odata, data, sizeof(float2)*WORK_SIZE, hipMemcpyDeviceToHost));

	for (i = 0; i < WORK_SIZE; i++)
		printf("Input value: %f, device output: %f\n", idata[i].x, hipCabsf(odata[i]));

	CUDA_CHECK_RETURN(hipFree((int*) d));
	CUDA_CHECK_RETURN(hipDeviceReset());
	hipFree(data);
	hipfftDestroy(plan);

	return 0;
}
