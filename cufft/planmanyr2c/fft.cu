#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_complex.h>


static const int WORK_SIZE = 10;

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }

int main(void) {
	float *d = NULL;
	int i;
	float idata[WORK_SIZE];
	float2 *odata_d=NULL;
	float2 *odata_h=(float2*)malloc((WORK_SIZE/2 +1) * sizeof(float2));

	for (i = 0; i < WORK_SIZE; i++){
		idata[i] = i;
	}
		hipfftReal a;
		hipfftHandle plan;
		float *data;
		hipMalloc((void**)&data, sizeof(float)*WORK_SIZE);
		hipMalloc((void**)&odata_d, sizeof(float2)*(WORK_SIZE/2+1));
		hipMemcpy(data,idata,sizeof(float)*WORK_SIZE,hipMemcpyHostToDevice);
		hipfftPlan1d(&plan, WORK_SIZE, HIPFFT_R2C,1);
		//cufftPlanMany(cufftHandle *plan, int rank, int *n, int *inembed, int istride, int idist, int *onembed, int ostride, int odist, cufftType type, int batch);

		int rank = 1;
		int n[1];
		int inembed[1];
		int istride=1;
		int idist=5;
		int onembed[1];
		int ostride = 1;
		int odist = 5;
		int batch =2;

		n[0]=WORK_SIZE/2;

		inembed[0]=WORK_SIZE;

		onembed[0]=WORK_SIZE/2 + 1;


		//cufftPlanMany(&plan, rank, n, inembed, istride, idist, onembed, ostride, odist, CUFFT_R2C, batch);

		hipfftExecR2C(plan, data, odata_d);

		hipDeviceSynchronize();



	CUDA_CHECK_RETURN(hipMemcpy(odata_h, odata_d, sizeof(float2)*(WORK_SIZE/2+1), hipMemcpyDeviceToHost));

	for (i = 0; i < WORK_SIZE/2 + 1; i++)
		printf("%f\n",hipCabsf(odata_h[i]));

	CUDA_CHECK_RETURN(hipFree((int*) d));
	CUDA_CHECK_RETURN(hipDeviceReset());
	hipFree(data);
	hipFree(odata_d);
	free(odata_h);
	free(idata);
	hipfftDestroy(plan);

	return 0;
}

