#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_complex.h>


static const int WORK_SIZE = 10;

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }

int main(void) {
	int *d = NULL;
	int i;
	float2 idata[WORK_SIZE];
	float2 odata[WORK_SIZE];

	for (i = 0; i < WORK_SIZE; i++){
		idata[i].x = i;
		idata[i].y = 0;
	}
		hipfftReal a;
		hipfftHandle plan;
		hipfftComplex *data;
		hipMalloc((void**)&data, sizeof(float2)*WORK_SIZE);
		hipMemcpy(data,idata,sizeof(float2)*WORK_SIZE,hipMemcpyHostToDevice);
		//cufftPlan1d(&plan, WORK_SIZE, CUFFT_C2C,1);
		//cufftPlanMany(cufftHandle *plan, int rank, int *n, int *inembed, int istride, int idist, int *onembed, int ostride, int odist, cufftType type, int batch);

		int rank = 1;
		int n[1];
		int inembed[1];
		int istride=1;
		int idist=5;
		int onembed[1];
		int ostride = 1;
		int odist = 5;
		int batch =2;

		n[0]=WORK_SIZE/2;

		inembed[0]=WORK_SIZE;

		onembed[0]=WORK_SIZE;


		hipfftPlanMany(&plan, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, batch);

		hipfftExecC2C(plan, data, data, HIPFFT_FORWARD);

		hipDeviceSynchronize();



	CUDA_CHECK_RETURN(hipMemcpy(odata, data, sizeof(float2)*WORK_SIZE, hipMemcpyDeviceToHost));

	for (i = 0; i < WORK_SIZE; i++)
		printf("%f\n",hipCabsf(odata[i]));

	CUDA_CHECK_RETURN(hipFree((int*) d));
	CUDA_CHECK_RETURN(hipDeviceReset());
	hipFree(data);
	hipfftDestroy(plan);

	return 0;
}

