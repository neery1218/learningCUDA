#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <mex.h>
#include <hipfft/hipfft.h>
#include <hip/hip_complex.h>
#include <math.h>
__constant__ int size; 
__global__ void sum (double *array){
	array[blockDim.x * blockIdx.x + threadIdx.x]*=2; 
}
void mexFunction( int nlhs, mxArray *plhs[],
                  int nrhs, const mxArray *prhs[])
{              
    int size;              
	size = mxGetN(prhs[0]);

	double *x_h = mxGetPr(prhs[0]);
	double *x_d_1;
	double *x_d_2;  
      
    /* check for proper number of arguments */
    	if(nrhs!=1) {
        	mexErrMsgIdAndTxt("MyToolbox:arrayProduct:nrhs","Two inputs required.");
    	}
	int num_devices;
	hipGetDeviceCount(&num_devices);
	mexPrintf("num devices: %d",num_devices); 
	int offset = 4; 

	hipSetDevice(0);
	hipMalloc(&x_d_1, sizeof(double) *offset);
	hipMemcpy(x_d_1, x_h, sizeof(double)*offset, hipMemcpyHostToDevice);

	hipSetDevice(1); 
    	hipMalloc(&x_d_2, sizeof(double) * (size-offset));
	hipMemcpy(x_d_2, x_h+offset, sizeof(double)*(size-offset), hipMemcpyHostToDevice);

	hipSetDevice(0); 
	sum<<<1,offset>>>(x_d_1);

	hipSetDevice(1);
	sum<<<1,size-offset>>>(x_d_2); 

	plhs[0] = mxCreateDoubleMatrix(1,(mwSize)size,mxREAL);
	hipMemcpy(mxGetPr(plhs[0]), x_d_1, sizeof(double)*offset, hipMemcpyDeviceToHost);

	hipMemcpy(mxGetPr(plhs[0])+offset, x_d_2, sizeof(double)*(size-offset), hipMemcpyDeviceToHost);

	//free(x_h);
	hipFree(x_d_1);
	hipFree(x_d_2);  


    

   
}    
        
