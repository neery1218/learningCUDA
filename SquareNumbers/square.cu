
#include <hip/hip_runtime.h>
__global__ void cu_square(int* a)
{
     int idx = blockIdx.x * blockDim.x + threadIdx.x;
        a[idx]*=a[idx];
}
