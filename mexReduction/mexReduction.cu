#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <mex.h>
#include <hipfft/hipfft.h>
#include <hip/hip_complex.h>
#include <math.h>
__constant__ int size; 
__global__ void sum (double *array){
	extern __shared__ double shared_data[];

	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	shared_data[tid] = array[tid];

	__syncthreads(); 

	for (unsigned int s=blockDim.x/2; s>0; s/=2) {
		if (tid < s) {
			shared_data[tid] += shared_data[tid + s];
		}
		__syncthreads();
	}
	array[tid] = shared_data[tid]; 
	//array[tid] = 7; 
}
void mexFunction( int nlhs, mxArray *plhs[],
                  int nrhs, const mxArray *prhs[])
{              
    int size;              
	size = mxGetN(prhs[0]);

	double *x_h = mxGetPr(prhs[0]);
	double *x_d; 
      
    /* check for proper number of arguments */
    	if(nrhs!=1) {
        	mexErrMsgIdAndTxt("MyToolbox:arrayProduct:nrhs","Two inputs required.");
    	}

    	hipMalloc(&x_d, sizeof(double) * size);
	hipMemcpy(x_d, x_h, sizeof(double)*size, hipMemcpyHostToDevice);

	sum<<<1,8,sizeof(double)*8>>>(x_d);

	hipDeviceSynchronize(); 
	plhs[0] = mxCreateDoubleMatrix(1,(mwSize)size,mxREAL);

	hipMemcpy(mxGetPr(plhs[0]), x_d, sizeof(double)*size, hipMemcpyDeviceToHost);

	//free(x_h);
	hipFree(x_d); 


    

   
}    
        
