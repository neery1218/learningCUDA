
#include <hip/hip_runtime.h>
//make sure numbers above  match the matlab script

__constant__ int num_row;
__constant__ int num_col; 

__global__ void matrix_addition (int* a, int* b, int* c)//each block calculates a row
{
	

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	//c[x * num_col + y] = a[x * num_col + y] + b[x * num_col + y];
	c[y * num_row + x] = a[y * num_row + x] + b[y * num_row + x];

	
	
}
